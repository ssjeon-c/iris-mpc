
#include <hip/hip_runtime.h>
#define P 65519

extern "C" __global__ void matmul(int *c, unsigned short *output, unsigned int *a0Sums, unsigned int *a1Sums, int *b0Sums, int *b1Sums, size_t numRows, size_t numElements, size_t numCols, long long lCoeff, unsigned short *rngMasks0, unsigned short *rngMasks1)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements)
    {
        unsigned int a0s = a0Sums[idx % numRows];
        unsigned int a1s = a1Sums[idx % numRows];

        // Correct the sum to unsigned
        int b0s = b0Sums[idx / numRows] + numCols * 128;
        int b1s = b1Sums[idx / numRows] + numCols * 128;

        // Correct the intermediate results to unsigned
        long long c00 = c[idx] + ((a0s + b0s) << 7) - (numCols * 16384);
        long long c01 = c[idx + numElements] + ((a0s + b1s) << 7) - (numCols * 16384);
        long long c10 = c[idx + numElements * 2] + ((a1s + b0s) << 7) - (numCols * 16384);
        long long c11 = c[idx + numElements * 3] + ((a1s + b1s) << 7) - (numCols * 16384);
        unsigned short result = (((c00 + ((c01 + c10) << 8) + (c11 << 16))) * lCoeff) % P;

        output[idx] = ((unsigned int)P + (unsigned int)result + (unsigned int)rngMasks0[idx] - (unsigned int)rngMasks1[idx]) % (unsigned int)P;
    }
}

extern "C" __global__ void reconstructDistance(unsigned short *codes_result1, unsigned short *codes_result2, unsigned short *codes_result3, unsigned short *masks_result1, unsigned short *masks_result2, unsigned short *masks_result3, float *output, size_t numElements)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements)
    {
        short nom = ((unsigned int)codes_result1[idx] + (unsigned int)codes_result2[idx] + (unsigned int)codes_result3[idx]) % (unsigned int)P;
        short den = ((unsigned int)masks_result1[idx] + (unsigned int)masks_result2[idx] + (unsigned int)masks_result3[idx]) % (unsigned int)P;
        output[idx] = (((float)nom / (float)den)-1.0f) * (-0.5f);
    }
}